#include "hip/hip_runtime.h"
/* -*-c++-*- */
/** \file cuda_convolution.cu

    CUDA convolution implementation.

    Copyright (C) 2010 Bastian Goldluecke,
    <first name>AT<last name>.net
    
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
   
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>

#include "cuda_convolutions.h"
#include "cuda_convolutions.cuh"
#include "cuda_interface.h"
#include "cuda_helper.h"

#include "../defs.h"

/********************************************************
  Convolution kernel structure
*********************************************************/

// Init arbitrary convolution kernel
coco::cuda_kernel *coco::cuda_kernel_alloc( const gsl_matrix *m )
{
  cuda_kernel *k = new cuda_kernel;
  k->_w = m->size2;
  k->_h = m->size1;
  k->_separable = false;
  CUDA_SAFE_CALL( hipMalloc( &k->_data, k->_w * k->_h * sizeof(float) ));
  cuda_memcpy( k->_data, m );
  k->_data_x = NULL;
  k->_data_y = NULL;
  return k;
}


// Init separable convolution kernel
coco::cuda_kernel *coco::cuda_kernel_alloc_separable( const gsl_vector *vx, const gsl_vector *vy )
{
  cuda_kernel *k = new cuda_kernel;
  k->_w = vx->size;
  k->_h = vy->size;
  k->_separable = true;
  k->_data = NULL;
  CUDA_SAFE_CALL( hipMalloc( &k->_data_x, k->_w * sizeof(float) ));
  CUDA_SAFE_CALL( hipMalloc( &k->_data_y, k->_h * sizeof(float) ));
  cuda_memcpy( k->_data_x, vx );
  cuda_memcpy( k->_data_y, vy );
  return k;
}

// Release convolution kernel
void coco::cuda_kernel_free( cuda_kernel *k )
{
  if ( k==NULL ) {
    return;
  }
  if ( k->_data != NULL ) {
    CUDA_SAFE_CALL( hipFree( k->_data ));
  }
  if ( k->_data_x != NULL ) {
    CUDA_SAFE_CALL( hipFree( k->_data_x ));
  }
  if ( k->_data_y != NULL ) {
    CUDA_SAFE_CALL( hipFree( k->_data_y ));
  }
  delete k;
}


/**********************************************************
  Convolution functions (adapted from nVidia SDK examples)
***********************************************************/

////////////////////////////////////////////////////////////////////////////////
// Convolution configuration
// Size of tiles (blocks) for convolution operations
// Larger block sizes = less overhead for apron
////////////////////////////////////////////////////////////////////////////////

// Assuming ROW_TILE_W, KERNEL_RADIUS_ALIGNED and dataW 
// are multiples of coalescing granularity size,
// all global memory operations are coalesced in convolutionRowGPU()
#define ROW_TILE_W 128

// Assuming COLUMN_TILE_W and dataW are multiples
// of coalescing granularity size, all global memory operations 
// are coalesced in convolutionColumnGPU()
#define COLUMN_TILE_W 16
#define COLUMN_TILE_H 48


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter (from nVidia SDK)
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU( float *d_Result, const float *d_Data, const float *d_Kernel,
				   int KERNEL_RADIUS,
				   int KERNEL_RADIUS_ALIGNED,
				   int dataW,
				   int dataH )
{
  // Data cache
  extern __shared__ float data[];

  //Current tile and apron limits, relative to row start
  const int         tileStart = IMUL(blockIdx.x, ROW_TILE_W);
  const int           tileEnd = tileStart + ROW_TILE_W - 1;
  const int        apronStart = tileStart - KERNEL_RADIUS;
  const int          apronEnd = tileEnd   + KERNEL_RADIUS;

  //Clamp tile and apron limits by image borders
  const int    tileEndClamped = min(tileEnd, dataW - 1);
  const int apronStartClamped = max(apronStart, 0);
  const int   apronEndClamped = min(apronEnd, dataW - 1);

  //Row start index in d_Data[]
  const int          rowStart = IMUL(blockIdx.y, dataW);

  //Aligned apron start. Assuming dataW and ROW_TILE_W are multiples 
  //of half-warp size, rowStart + apronStartAligned is also a 
  //multiple of half-warp size, thus having proper alignment 
  //for coalesced d_Data[] read.
  const int apronStartAligned = tileStart - KERNEL_RADIUS_ALIGNED;
  
  const int loadPos = apronStartAligned + threadIdx.x;
  //Set the entire data cache contents
  //Load global memory values, if indices are within the image borders,
  //or initialize with zeroes otherwise
  if(loadPos >= apronStart){
    const int smemPos = loadPos - apronStart;
    
    data[smemPos] = 
      (loadPos < apronStartClamped) ? d_Data[rowStart + apronStartClamped] : 
      ( (loadPos > apronEndClamped) ? d_Data[rowStart + apronEndClamped] :
	d_Data[rowStart + loadPos] );
  }

  //Ensure the completness of the loading stage
  //because results, emitted by each thread depend on the data,
  //loaded by another threads
  __syncthreads();

  const int writePos = tileStart + threadIdx.x;
  //Assuming dataW and ROW_TILE_W are multiples of half-warp size,
  //rowStart + tileStart is also a multiple of half-warp size,
  //thus having proper alignment for coalesced d_Result[] write.
  if(writePos <= tileEndClamped){
    const int smemPos = writePos - apronStart;
    float sum = 0;
    for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++) {
      sum += data[smemPos + k] * d_Kernel[KERNEL_RADIUS - k];
    }
    d_Result[rowStart + writePos] = sum;
  }
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter (from nVidia SDK)
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnGPU(
    float *d_Result,
    const float *d_Data,
    const float *d_Kernel,
    int KERNEL_RADIUS,
    int dataW,
    int dataH,
    int smemStride,
    int gmemStride )
{
  //Data cache
  extern __shared__ float data[];

  //Current tile and apron limits, in rows
  const int         tileStart = IMUL(blockIdx.y, COLUMN_TILE_H);
  const int           tileEnd = tileStart + COLUMN_TILE_H - 1;
  const int        apronStart = tileStart - KERNEL_RADIUS;
  const int          apronEnd = tileEnd   + KERNEL_RADIUS;

  //Clamp tile and apron limits by image borders
  const int    tileEndClamped = min(tileEnd, dataH - 1);
  const int apronStartClamped = max(apronStart, 0);
  const int   apronEndClamped = min(apronEnd, dataH - 1);

  //Current column index
  const int       columnStart = IMUL(blockIdx.x, COLUMN_TILE_W) + threadIdx.x;

  //Shared and global memory indices for current column
  int smemPos = IMUL(threadIdx.y, COLUMN_TILE_W) + threadIdx.x;
  int gmemPos = IMUL(apronStart + threadIdx.y, dataW) + columnStart;
  //Cycle through the entire data cache
  //Load global memory values, if indices are within the image borders,
  //or initialize with zero otherwise
  for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y) {
    data[smemPos] = (y < apronStartClamped) ? d_Data[IMUL(apronStartClamped,dataW) + columnStart] :
      ((y > apronEndClamped) ? d_Data[IMUL(apronEndClamped,dataW) + columnStart] :
       d_Data[gmemPos]);
    smemPos += smemStride;
    gmemPos += gmemStride;
  }

  //Ensure the completness of the loading stage
  //because results, emitted by each thread depend on the data, 
  //loaded by another threads
  __syncthreads();
  //Shared and global memory indices for current column
  smemPos = IMUL(threadIdx.y + KERNEL_RADIUS, COLUMN_TILE_W) + threadIdx.x;
  gmemPos = IMUL(tileStart + threadIdx.y , dataW) + columnStart;
  //Cycle through the tile body, clamped by image borders
  //Calculate and output the results
  for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y) {
    float sum = 0;
    for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++) {
      sum += 
	data[smemPos + IMUL(k, COLUMN_TILE_W)] *
	d_Kernel[KERNEL_RADIUS - k];
    }
    d_Result[gmemPos] = sum;
    smemPos += smemStride;
    gmemPos += gmemStride;
  }
}



__global__ void cuda_convolution_nonsep_device( int W, int H,
						float *k,
						int w, int h,
						int w2, int h2,
						const float *s, float *d )
{
  // Global thread index
  int ox = blockDim.x * blockIdx.x + threadIdx.x;
  int oy = blockDim.y * blockIdx.y + threadIdx.y;
  if ( ox>=W || oy>=H ) {
    return;
  }
  int o = oy*W + ox;

  // Compute local convolution
  float v = 0.0f;
  float n = 0.0f;
  int index=0;
  for ( int j=0; j<h; j++ ) {
    for ( int i=0; i<w; i++ ) {
      
      int xx = ox - w2 + i;
      int yy = oy - h2 + j;

      if ( xx>=0 && xx<W && yy>=0 && yy<H ) {
	float kv = k[index];
	n += kv;
	v += kv * s[ yy * W + xx ];
      }

      index++;
    }
  }

  if ( n>0.0f ) {
    v /= n;
  }

  d[o] = v;
}


// Slow nonseparable version
static bool cuda_convolution_nonsep( const coco::cuda_kernel *kernel, 
				     size_t W, size_t H,
				     const float* in, float *out )
{
  // Matrix size has to be multiple of block size.
  dim3 dimGrid, dimBlock;
  coco::cuda_default_grid( W,H, dimGrid, dimBlock );
  //coco::write_test_image_unsigned( W,H, (float*)in, "./out/conv_in.png", 0 );

  // Compute divergence step
  cuda_convolution_nonsep_device<<< dimGrid, dimBlock >>>
    ( W, H,
      kernel->_data,
      kernel->_w, kernel->_h,
      ( kernel->_w -1 ) / 2, ( kernel->_h - 1 ) / 2, 
      in, out );

  //coco::write_test_image_unsigned( W,H, out, "./out/conv_out.png", 0 );
  //assert( false );
  return true;
}




// Convolve array with kernel
bool coco::cuda_convolution( const cuda_kernel *kernel, 
			     size_t W, size_t H,
			     const float* in, float* out )
{
  if ( !kernel->_separable ) {
    return cuda_convolution_nonsep( kernel, W,H, in, out );
  }
  // Needs a temp array
  float *tmp = NULL;
  CUDA_SAFE_CALL( hipMalloc( &tmp, W*H*sizeof(float) ));

  // Compute radius
  const int KERNEL_RADIUS_X = kernel->_w / 2;
  assert( kernel->_w == size_t(KERNEL_RADIUS_X*2 + 1) );
  const int KERNEL_RADIUS_Y = kernel->_h / 2;
  assert( kernel->_h == size_t(KERNEL_RADIUS_Y*2 + 1) );
  // Compute alignment radius: must be multiple of 16 (half warp size)
  // for maximum performance.
  const int KERNEL_RADIUS_ALIGNED = ((KERNEL_RADIUS_X-1) / 16 + 1) * 16;

  // Call CUDA kernels
  dim3 blockGridRows(iDivUp(W, ROW_TILE_W), H);
  dim3 blockGridColumns(iDivUp(W, COLUMN_TILE_W), iDivUp(H, COLUMN_TILE_H));
  dim3 threadBlockRows(KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS_X);
  dim3 threadBlockColumns(COLUMN_TILE_W, 8);

  CUDA_SAFE_CALL( hipMemset( out, 0, sizeof(float)*W*H ));
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  size_t memsize_row = sizeof(float) * (KERNEL_RADIUS_X + ROW_TILE_W + KERNEL_RADIUS_X);
  convolutionRowGPU<<<blockGridRows, threadBlockRows, memsize_row>>>
    (tmp, in,
     kernel->_data_x,
     KERNEL_RADIUS_X,
     KERNEL_RADIUS_ALIGNED,
     W,H );
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  size_t memsize_column = sizeof(float) * COLUMN_TILE_W * (KERNEL_RADIUS_Y + COLUMN_TILE_H + KERNEL_RADIUS_Y);
  convolutionColumnGPU<<<blockGridColumns, threadBlockColumns, memsize_column>>>
    ( out, tmp,
      kernel->_data_y, KERNEL_RADIUS_Y,
      W,H,
      COLUMN_TILE_W * threadBlockColumns.y,
      W * threadBlockColumns.y );

  CUDA_SAFE_CALL( hipFree( tmp ));

  return false;
}



static __global__ void convolution_row3_device( int W, int H,
						float k0, float k1, float k2,
						const float *in, float *out )
{
  // Global thread index
  const int ox = IMUL( blockDim.x, blockIdx.x ) + threadIdx.x;
  const int oy = IMUL( blockDim.y, blockIdx.y ) + threadIdx.y;
  if ( ox >= W || oy >= H ) {
    return;
  }
  const int o = IMUL( oy,W ) + ox;
  if ( ox==0 ) {
    out[o] = (k2 * in[o+1] + k1 * in[o]) / (k1+k2);
  }
  else if ( ox==W-1 ) {
    out[o] = (k1 * in[o] + k0 * in[o-1]) / (k0+k1);
  }
  else {
    out[o] = k2 * in[o+1] + k0 * in[o-1] + k1 * in[o];
  }
}


static __global__ void convolution_column3_device( int W, int H,
						   float k0, float k1, float k2,
						   const float *in, float *out )
{
  // Global thread index
  const int ox = IMUL( blockDim.x, blockIdx.x ) + threadIdx.x;
  const int oy = IMUL( blockDim.y, blockIdx.y ) + threadIdx.y;
  if ( ox >= W || oy >= H ) {
    return;
  }
  const int o = IMUL( oy,W ) + ox;
  if ( oy==0 ) {
    out[o] = (k2 * in[o+W] + k1 * in[o]) / (k1+k2);
  }
  else if ( oy==H-1 ) {
    out[o] = (k1 * in[o] + k0 * in[o-W]) / (k0+k1);
  }
  else {
    out[o] = k2 * in[o+W] + k0 * in[o-W] + k1 * in[o];
  }
}


// Fast convolution for Row-3 kernel
bool coco::cuda_convolution_row( float k0, float k1, float k2,
				 size_t W, size_t H,
				 const float* in, float* out )
{
  dim3 dimBlock;
  dim3 dimGrid;
  cuda_default_grid( W,H, dimGrid, dimBlock );
  convolution_row3_device<<< dimGrid, dimBlock >>>
    ( W,H, k0,k1,k2,
      in, out );
  return true;
}


// Fast convolution for Column-3 kernel
bool coco::cuda_convolution_column( float k0, float k1, float k2,
				    size_t W, size_t H,
				    const float* in, float* out )
{
  dim3 dimBlock;
  dim3 dimGrid;
  cuda_default_grid( W,H, dimGrid, dimBlock );
  convolution_column3_device<<< dimGrid, dimBlock >>>
    ( W,H, k0,k1,k2,
      in, out );
  return true;
}


