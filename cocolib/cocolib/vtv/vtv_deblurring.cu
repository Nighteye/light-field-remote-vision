#include "hip/hip_runtime.h"
/* -*-c++-*- */
#include <iostream>

#include "vtv.h"
#include "vtv.cuh"

#include "../common/gsl_matrix_helper.h"
#include "../cuda/cuda_helper.h"
#include "../cuda/cuda_kernels.cuh"
#include "../defs.h"



// Perform one primal step (several iterations of gradient descent for
// the prox operator
bool coco::coco_vtv_deblurring_primal_step( coco_vtv_data *data )
{
  size_t W = data->_W;
  size_t H = data->_H;
  coco_vtv_workspace *w = data->_workspace;

  // Compute divergence step
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    tv_primal_descent_step_device<<< w->_dimGrid, w->_dimBlock >>>
      ( W,H, data->_tau,
	w->_U[i], w->_Uq[i], w->_X1[i], w->_X2[i] );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
  }

  // Compute deblurring step (second dual variable)
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    // bq*q gives step size
    cuda_convolution( w->_bq, data->_W, data->_H,
		      w->_Uq[i], w->_temp[i] );
    // add to solution
    cuda_add_scaled_to_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_temp[i], -data->_tau, w->_Uq[i] );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
  }

  return true;
}



// Perform one dual step
bool coco::coco_vtv_deblurring_dual_step( coco_vtv_data *data )
{
  // First variables xi
  coco_vtv_rof_dual_step( data );
  // Then variable eta
  size_t W = data->_W;
  size_t H = data->_H;
  coco_vtv_workspace *w = data->_workspace;

  // Kernel call for each channel
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    // b*u
    cuda_convolution( w->_b, W,H,
		      w->_Uq[i], w->_temp[i] );
    // b*u-f
    cuda_subtract_from_device<<< w->_dimGrid, w->_dimBlock >>>
      ( W,H, w->_F[i], w->_temp[i] );
    // update step
    cuda_add_scaled_to_device<<< w->_dimGrid, w->_dimBlock >>>
      ( W,H, w->_temp[i], data->_sigma, w->_temp[i] );
    // prox step (for q) - depends on data term regularizer
    switch ( data->_data_term_p ) {
    case 1:
      {
	cuda_reproject_to_ball_1d<<< w->_dimGrid, w->_dimBlock >>>
	  ( data->_W, data->_H,
	    1.0 / (2.0 * data->_lambda),
	    w->_temp[i] );
      }
      break;
    case 2:
      {
	cuda_scale_device<<< w->_dimGrid, w->_dimBlock >>>
	  ( data->_W, data->_H, w->_temp[i], 1.0 / ( 1.0 + data->_sigma * data->_lambda ) );
      }
      break;
    default:
      {
	ERROR( "Data term regularizer only supports p=1 or p=2." << std::endl );
	assert( false );
      }
    }

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
  }

  return true;
}


// Perform one single primal descent step
bool coco::coco_vtv_deblurring_primal_descent_step( coco_vtv_data *data )
{
  size_t W = data->_W;
  size_t H = data->_H;
  coco_vtv_workspace *w = data->_workspace;

  // Start descent from current solution
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    CUDA_SAFE_CALL( hipMemcpy( w->_Uq[i], w->_U[i], w->_nfbytes, hipMemcpyDeviceToDevice ));
  }
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  // Extra buffer (location where prox operator is evaluated)
  float *V = NULL;
  CUDA_SAFE_CALL( hipMalloc( &V, w->_nfbytes ));

  // Iterate over channels
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    // Compute location where to evaluate the prox operator
    tv_primal_descent_step_device<<< w->_dimGrid, w->_dimBlock >>>
      ( W,H, data->_tau,
	w->_Uq[i], V, w->_X1[i], w->_X2[i] );

    // b*u
    cuda_convolution( w->_b, data->_W, data->_H,
		      w->_Uq[i], w->_temp[i] );
    // b*u-f
    cuda_subtract_from_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_F[i], w->_temp[i] );
    // bq * (b*u-f)
    cuda_convolution( w->_bq, data->_W, data->_H,
		      w->_temp[i], w->_G[i] );
    // 1/lambda * ...
    cuda_scale_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_G[i], 1.0 / data->_lambda );
    // + u / tau
    cuda_add_scaled_to_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_Uq[i], 1.0 / data->_tau, w->_G[i] );
    // - v / tau
    cuda_add_scaled_to_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, V, -1.0 / data->_tau, w->_G[i] );
    
    // Gradient step
    CUDA_SAFE_CALL( hipMemcpy( w->_Uq[i], V, w->_nfbytes, hipMemcpyDeviceToDevice ));
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    cuda_add_scaled_to_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_G[i], -data->_tau, w->_Uq[i] );
    
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
  }

  return true;
}





// Perform one single shrinkage step (ISTA)
bool coco::coco_vtv_deblurring_ista_step( coco_vtv_data *data )
{
  size_t W = data->_W;
  size_t H = data->_H;
  coco_vtv_workspace *w = data->_workspace;

  // Start descent from current solution
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    CUDA_SAFE_CALL( hipMemcpy( w->_Uq[i], w->_U[i], w->_nfbytes, hipMemcpyDeviceToDevice ));
  }
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  // Compute gradient of data term
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    // b*u
    cuda_convolution( w->_b, W,H,
		      w->_Uq[i], w->_temp[i] );

    // b*u-f
    cuda_subtract_from_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_F[i], w->_temp[i] );
    // bq * (b*u-f)
    cuda_convolution( w->_bq, data->_W, data->_H,
		      w->_temp[i], w->_G[i] );
    // 1/L * ...
    cuda_scale_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_G[i], -1.0 / ( data->_lambda * data->_L ));

    // Add current solution
    cuda_add_to_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_Uq[i], w->_G[i] );
  }


  // Perform a number of primal/dual ROF iterations
  data->_tau = 0.3 / sqrt( 8.0 );
  data->_sigma = 0.3 / sqrt( 8.0 );
  const size_t iter_rof = data->_inner_iterations;

  for ( size_t k=0; k<iter_rof; k++ ) {
    coco_vtv_rof_dual_step( data );

    // Primal step kernel call for each channel
    for ( size_t i=0; i<data->_nchannels; i++ ) {
      cuda_rof_primal_prox_step_device<<< w->_dimGrid, w->_dimBlock >>>
	( data->_W, data->_H, data->_tau, 1.0 / data->_L,
	  w->_Uq[i], w->_Uq[i], w->_G[i], w->_X1[i], w->_X2[i] );
      CUDA_SAFE_CALL( hipDeviceSynchronize() );
    }
  }

  return true;
}



// Compute primal energy
double coco::coco_vtv_deblurring_primal_energy( coco_vtv_data *data )
{
  size_t W = data->_W;
  size_t H = data->_H;
  coco_vtv_workspace *w = data->_workspace;

  // Compute gradient of current solution
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    cuda_compute_gradient_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_U[i], w->_X1q[i], w->_X2q[i] );
  }
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  // Compute largest singular value of gradient matrix
  cuda_compute_largest_singular_value_device<<< w->_dimGrid, w->_dimBlock >>>
    ( data->_W, data->_H, 
      w->_X1q[0], w->_X2q[0], w->_X1q[1], w->_X2q[1], w->_X1q[2], w->_X2q[2],
      w->_G[0] );

  // Compute gradient of data term
  for ( size_t i=0; i<data->_nchannels; i++ ) {
    // b*u
    cuda_convolution( w->_b, data->_W, data->_H,
		      w->_Uq[i], w->_temp[i] );
    // b*u-f
    cuda_subtract_from_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_F[i], w->_temp[i] );
    // square
    cuda_square_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_temp[i] );
    // 1/(2 lambda) * ...
    cuda_scale_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_temp[i], 1.0 / ( 2.0 * data->_lambda ));

    // Add to smoothness term
    cuda_add_to_device<<< w->_dimGrid, w->_dimBlock >>>
      ( data->_W, data->_H, w->_temp[i], w->_G[0] );
  }
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  // Copy temp array to CPU
  cuflt *E = new cuflt[ W*H ];
  CUDA_SAFE_CALL( hipMemcpy( E, w->_G[0], w->_nfbytes, hipMemcpyDeviceToHost ));
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  // Compute total energy
  double e = 0.0;
  for ( size_t i=0; i<W*H; i++ ) {
    e += E[i];
  }
  delete[] E;
  return e / double(W*H);
}




// Perform one iteration of Algorithm 1, Chambolle-Pock
bool coco::coco_vtv_deblurring_iteration_chambolle_pock_1( coco_vtv_data *data )
{
  // Todo: verify correct maximum step sizes.
  data->_tau = 0.3 / sqrt( 8.0 );
  data->_sigma = 0.3 / sqrt( 8.0 );
  bool ok2 = coco_vtv_deblurring_dual_step( data );
  bool ok1 = coco_vtv_deblurring_primal_step( data );
  bool ok3 = coco_vtv_rof_overrelaxation( data, 1.0 );
  return ok1 && ok2 && ok3;
}


// Perform one iteration of Algorithm 1, Chambolle-Pock
bool coco::coco_vtv_deblurring_iteration_chambolle_pock_2( coco_vtv_data *data )
{
  // Todo: verify correct maximum step sizes.
  bool ok2 = coco_vtv_deblurring_dual_step( data );
  bool ok1 = coco_vtv_deblurring_primal_step( data );

  data->_gamma = 1.0 / data->_lambda;
  cuflt theta = 1.0 / sqrt( 1.0 + 2.0 * data->_gamma * data->_tau );
  //data->_tau = theta * data->_tau;
  //data->_sigma = data->_sigma / theta;
  data->_tau = data->_tau * theta;
  data->_sigma = data->_sigma / theta;
  //cout << "new tau: " << data->_tau << "  sigma: " << data->_sigma << "  gamma: " << data->_gamma << "  theta: " << theta << std::endl;
  bool ok3 = coco_vtv_rof_overrelaxation( data, theta );
  return ok1 && ok2 && ok3;
}



// Perform one iteration of Algorithm 1, Chambolle-Pock
bool coco::coco_vtv_deblurring_iteration_arrow_hurwicz( coco_vtv_data *data )
{
  // Todo: verify correct maximum step sizes.
  data->_tau = 0.1 * data->_lambda;
  data->_sigma = 0.3 / sqrt( 8.0 );
  bool ok2 = coco_vtv_rof_dual_step( data );
  //  bool ok1 = coco_vtv_deblurring_primal_step( data );
  bool ok1 = coco_vtv_deblurring_primal_descent_step( data );
  bool ok3 = coco_vtv_rof_overrelaxation( data, 0.0 );
  return ok1 && ok2 && ok3;
}



// Perform one iteration of Algorithm 1, Chambolle-Pock
bool coco::coco_vtv_deblurring_iteration_fista( coco_vtv_data *data )
{
  // Todo: verify correct maximum step sizes.
  data->_tau = 0.3 / sqrt( 8.0 );
  data->_sigma = 0.3 / sqrt( 8.0 );
  data->_L = 1.0 / data->_lambda;
  bool ok2 = coco_vtv_deblurring_ista_step( data );
  cuflt alpha_new = 0.5 * ( 1.0 + sqrt( 1.0 + 4.0 * pow( data->_alpha, 2.0 ) ));
  bool ok3 = coco_vtv_rof_overrelaxation( data, ( data->_alpha - 1.0 ) / alpha_new );
  data->_alpha = alpha_new;
  return ok2 && ok3;
}


