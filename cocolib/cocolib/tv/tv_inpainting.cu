#include "hip/hip_runtime.h"
/* -*-c++-*- */
/** \file tv_inpainting.cu
   Algorithms to solve the TV model with inpainting data term.

   Workspace handling and access code.

   Copyright (C) 2010 Bastian Goldluecke,
                      <first name>AT<last name>.net

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include "tv_inpainting.h"
#include "tv_inpainting.cuh"

#include "../defs.h"
#include "../cuda/cuda_helper.h"


// Compute gradient of inpainting data term (device)
__global__ void compute_inpainting_gradient_device( int W, int H,
						    float *dE, double lambda,
						    float *mask, float *u, float *f )
{
  // Global thread index
  int ox = blockDim.x * blockIdx.x + threadIdx.x;
  int oy = blockDim.y * blockIdx.y + threadIdx.y;
  if ( ox>=W || oy>=H ) {
    return;
  }
  int o = oy*W + ox;
  dE[o] = mask[o] * ( u[o] - f[o] ) / lambda;
}

// Compute gradient of inpainting data term (device code wrapper)
static bool compute_inpainting_gradient( void* context,
					 size_t W, size_t H,
					 float *u, float *dE )
{
  // Recover workspace
  coco::tv_inpainting_workspace *w = (coco::tv_inpainting_workspace*)context;

  // Kernel call for each channel
  compute_inpainting_gradient_device<<< w->_dimGrid, w->_dimBlock >>>
    ( W, H,
      dE, w->_lambda, w->_mask, u, w->_f );

  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  return true;
}


// Alloc PDE data with sensible defaults
coco::tv_inpainting_data* coco::tv_inpainting_data_alloc( gsl_matrix *mask, gsl_matrix *f )
{
  tv_inpainting_data *data = new tv_inpainting_data;
  size_t W = f->size2;
  size_t H = f->size1;

  // Workspace
  data->_workspace = new tv_inpainting_workspace;
  memset( data->_workspace, 0, sizeof( tv_inpainting_workspace ));
  tv_inpainting_workspace *w = data->_workspace;

  // CUDA Block dimensions
  w->_dimBlock = dim3( cuda_default_block_size_x(),
		       cuda_default_block_size_y() );
  size_t blocks_w = W / w->_dimBlock.x;
  if ( W % w->_dimBlock.x != 0 ) {
    blocks_w += 1;
  }
  size_t blocks_h = H / w->_dimBlock.y;
  if ( H % w->_dimBlock.y != 0 ) {
    blocks_h += 1;
  }
  w->_dimGrid = dim3(blocks_w, blocks_h);

  // Arrays for f and k*f
  size_t Nf = W*H*sizeof(float);
  CUDA_SAFE_CALL( hipMalloc( &w->_f, Nf ));
  CUDA_SAFE_CALL( hipMalloc( &w->_mask, Nf ));

  // Copy f
  cuda_memcpy( w->_f, f );
  // Copy mask
  cuda_memcpy( w->_mask, mask );

  // TV-Convex optimizer
  data->_lambda = 0.01;
  data->_tv_convex = tv_convex_data_alloc( f->size2, f->size1,
					   &compute_inpainting_gradient,
					   w );
  return data;
}



// Free up PDE data
bool coco::tv_inpainting_data_free( tv_inpainting_data *data )
{
  // Free GPU fields
  tv_inpainting_workspace *w = data->_workspace;
  CUDA_SAFE_CALL( hipFree( w->_f ));
  CUDA_SAFE_CALL( hipFree( w->_mask ));
  tv_convex_data_free( data->_tv_convex );
  delete data->_workspace;
  delete data;
  return true;
}



// Initialize workspace with current solution
bool coco::tv_inpainting_initialize( tv_inpainting_data *data,
				     gsl_matrix* u )
{
  data->_workspace->_lambda = data->_lambda;
  data->_tv_convex->_L = 1.0 / data->_lambda;
  return tv_convex_initialize( data->_tv_convex, u );
}

// Get current solution
bool coco::tv_inpainting_get_solution( tv_inpainting_data *data,
				       gsl_matrix* u )
{
  return tv_convex_get_solution( data->_tv_convex, u );
}


double coco::tv_inpainting_energy( tv_inpainting_data *data )
{
  return tv_convex_energy( data->_tv_convex );
}


/*****************************************************************************
       TV-Inpainting algorithm I: Specialized FISTA (Beck/Teboulle 2008)
*****************************************************************************/

// Perform one full iteration
bool coco::tv_inpainting_iteration_fista( tv_inpainting_data *data )
{
  return tv_convex_iteration_fista( data->_tv_convex );
}

