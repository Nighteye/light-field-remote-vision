#include "hip/hip_runtime.h"
/* -*-c++-*- */
/** \file tc_deblurring_fista.cu
   Algorithms to solve the TC model with linear data term.

   Copyright (C) 2011 Bastian Goldluecke,
                      <first name>AT<last name>.net

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include "tc_l2.h"
#include "tc_deblurring.h"
#include "tc.cuh"
#include "tc_arrays.cuh"

#include "../defs.h"
#include "../cuda/cuda_helper.h"
#include "../cuda/cuda_kernels.cuh"
#include "../common/gsl_image.h"
#include "../common/gsl_matrix_derivatives.h"
#include "../common/menger_curvature.h"
#include "../common/profiler.h"


/******************************************
    EXPERIMENTAL MINIMIZATION ALGORITHM
    see initial version paper ICCV 2011
*******************************************/


/******************************************
    MAIN ITERATION: Fista
******************************************/

__global__ void cvd_fista_init_derivative_device( int W, int H,
						  stcflt step,
						  stcflt *uq,
						  stcflt *mask, stcflt *f,
						  stcflt *D )
{
  // Global thread index
  int x0 = blockDim.x * blockIdx.x + threadIdx.x;
  int x1 = blockDim.y * blockIdx.y + threadIdx.y;
  if ( x0>=W || x1 >= H ) {
    return;
  }
  int o = x1*W + x0;
  stcflt uqv = uq[o];
  D[o] = uqv - step * f[o];
}


__global__ void cvd_fista_overrelaxation_device( int W, int H,
						 stcflt alpha,
						 stcflt *u,
						 stcflt *uq )

{
  // Global thread index
  int x0 = blockDim.x * blockIdx.x + threadIdx.x;
  int x1 = blockDim.y * blockIdx.y + threadIdx.y;
  if ( x0>=W || x1>=H ) {
    return;
  }
  int o = x1*W + x0;
  stcflt uo = u[o];
  stcflt un = uq[o];
  u[o] = un;
  uq[o] = uo + alpha * (un - uo);
}



// Perform one iteration (outer loop)
bool coco::tc_deblurring_fista_init( tc_data *data )
{
  tc_workspace *w = data->_workspace;
  w->_t = 1.0;
  w->_lambda = data->_lambda;
  w->_L = 1.0 / w->_lambda;
  w->_rof_lambda = 1.0 / w->_L;
  stcflt K = 3.0 * pow( w->_N, 2.0 ) * sqrt( 8.0 );
  w->_sigma = 2.0 / (K * K * w->_rof_lambda);

  // Start iteration with input f
  CUDA_SAFE_CALL( hipMemcpy( w->_u, w->_a, w->_Nf, hipMemcpyDeviceToDevice ));
  CUDA_SAFE_CALL( hipMemcpy( w->_uq, w->_a, w->_Nf, hipMemcpyDeviceToDevice ));
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  return true;
}


bool coco::tc_deblurring_fista_iteration( tc_data *data )
{
  tc_workspace *w = data->_workspace;
  data->_iteration ++;

  TRACE5( "****** FISTA ITERATION " << data->_iteration << " ******" << std::endl );

  // Step 1: Compute derivative of uq using inpainting mask and Lipschitz
  // constant
  // b*u
  cuda_convolution( w->_b, data->_W, data->_H,
		    w->_uq, w->_u_star );
  // b*u-f
  cuda_subtract_from_device<<< w->_dimGrid, w->_dimBlock >>>
    ( data->_W, data->_H, w->_a, w->_u_star );
  // bq * (b*u-f)
  cuda_convolution( w->_bq, data->_W, data->_H,
		    w->_u_star, w->_f );
  // 1/L * ...
  cuda_scale_device<<< w->_dimGrid, w->_dimBlock >>>
    ( data->_W, data->_H, w->_f, -1.0 / ( w->_lambda * w->_L ));
  
  // Add current solution
  cuda_add_to_device<<< w->_dimGrid, w->_dimBlock >>>
    ( data->_W, data->_H, w->_uq, w->_f );
  
  // Step 1a: Backup old u for overrelaxation
  CUDA_SAFE_CALL( hipMemcpy( w->_u_star, w->_u, w->_Nf, hipMemcpyDeviceToDevice ));
  CUDA_SAFE_CALL( hipDeviceSynchronize() );

  // Step 2: Perform inner ROF iterations
  TRACE6( "  ROF iterations [" );
  for ( size_t i=0; i<data->_inner_iterations; i++ ) {
    TRACE6( "." );

    // Primal step: Compute K^* xi - 1/lambda f
    if (!tc_l2_iteration( data )) {
      ERROR( "ROF iteration failure." << std::endl );
      return false;
    }
  }

  TRACE6( "]" << std::endl );
    
  // Step 3: Recover u and compute overrelaxation in uq
  double new_t = 0.5 * ( 1.0 + sqrt( 1.0 + 4.0 * pow( (double)w->_t, 2.0 )) );
  double alpha = (w->_t - 1.0) / new_t;
  cvd_fista_overrelaxation_device<<< w->_dimGrid, w->_dimBlock >>>
    ( data->_W, data->_H,
      alpha,
      w->_u_star,
      w->_uq );
  w->_t = new_t;

  return true;
}






